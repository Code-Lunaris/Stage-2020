
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 32
#define M 4

__global__ void reverse(int *tab, int n){

    __shared__ int s[N];
    int t = threadIdx.x;
    int tr = n-t-1;
    s[t] = tab[t];
    __syncthreads();
    tab[t] = s[tr];
}


int main (void){

    int *array;
    //int *result;

    int *d_array;


    int size = N*sizeof(int);

    // Alocate memory space for nvidia GPU
    hipMalloc((void **)&d_array, size);

    array = (int *)malloc(size);
    

    for(int i=0; i<N; i++){
        array[i] = i;
    }

    //copy inputs to device
    hipMemcpy(d_array, array, size, hipMemcpyHostToDevice);

    
    reverse<<<1, N>>>(d_array, N);

    hipMemcpyAsync(array, d_array, size, hipMemcpyDeviceToHost);

    for(int i=0; i<N; i++)
        printf("%d\n", array[i]);

    hipFree(d_array);

    free(array);

    return 0;
}