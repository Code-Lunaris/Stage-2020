
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <npp.h>

#define N 5 
#define M 2

__global__ void add(int *m1, int *m2, int *res, int n){
    //int index = threadIdx.x + blockIdx.x * blockDim.x;
    int ix = threadIdx.x+blockIdx.x*blockDim.x;

    if(ix<M*N) 
        res[ix] = m1[ix] + m2[ix];
}


int main (void){

    int *mtx1, *mtx2, *mtx_res;
    int *d_mtx1, *d_mtx2, *d_mtx_res;

    int size = M*N*sizeof(int);

    // Alocate memory space for nvidia GPU
    hipMalloc((void **)&d_mtx1, size);
    hipMalloc((void **)&d_mtx2, size);
    hipMalloc((void **)&d_mtx_res, size);

    mtx1 = (int *)malloc(size);
    mtx2 = (int *)malloc(size);
    mtx_res = (int *)malloc(size);

    for(int i=0; i<M; i++){
        for(int j=0; j<N; j++){
            mtx1[N*i+j] = i*N+j;
            mtx2[N*i+j] = i*N+j;
            //printf("%d (%d)\t", mtx1[N*i+j], N*i+j);
        }
        //printf("\n");
    }


    // //copy inputs to device
    hipMemcpy(d_mtx1, mtx1, size, hipMemcpyHostToDevice);
    hipMemcpy(d_mtx2, mtx2, size, hipMemcpyHostToDevice);

    
    add<<<N, M>>>(d_mtx2, d_mtx1, d_mtx_res, N);

    hipMemcpy(mtx_res, d_mtx_res, size, hipMemcpyDeviceToHost);

    for(int i=0; i<M; i++){
        for(int j=0; j<N; j++){
            printf("%d\t", mtx_res[N*i+j]);
        }
        printf("\n");
    }

    hipFree(d_mtx1);
    hipFree(d_mtx2);
    hipFree(d_mtx_res);

    free(mtx1);
    free(mtx2);
    free(mtx_res);

    return 0;
}